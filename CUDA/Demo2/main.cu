#include "hip/hip_runtime.h"
//include,system
#include<iostream>

#include<hip/hip_runtime.h>
#include <NTL/ZZ.h>

using namespace std;
using namespace NTL;
#include<time.h>
__global__ void vectorAdd(const float *A,const float *B,float *C,int numElements)
{
int i=blockDim.x*blockIdx.x+threadIdx.x;
if(i<numElements)
{
C[i]=A[i]+B[i];
}
}
int main(void)
{ // Print the vector length to be used, and compute its size
int numElements=5;
size_t size=numElements*sizeof(float);

// Allocate the host input vector A
float *h_A=(float *)malloc(size);
// Allocate the host input vector B
float *h_B = (float *)malloc(size);
// Allocate the host output vector C
float *h_C = (float *)malloc(size);
//Initialize the host input vectors
for(int i=0;i<numElements;++i)
{
h_A[i]=2;
h_B[i]=12;
}
//Allocate the device input vector A
float *d_A=NULL;
hipMalloc((void **)&d_A,size);
//Allocate the device input vector B
float *d_B=NULL;
hipMalloc((void **)&d_B,size);
//Allocate the device output vector C
float *d_C=NULL;
hipMalloc((void **)&d_C,size);
//Copy the host input vector A and B in host memory to
//the device input vectors in device memory

hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);
//Launch the Vector Add CUDA Kernel
int threadsPerBlock=numElements;
int blocksPerGrid=1;

vectorAdd<<<blocksPerGrid,threadsPerBlock>>>(d_A,d_B,d_C,numElements);
//return the output vector d_A in device to the host output vectors in host

hipMemcpy(h_C, d_C,size,hipMemcpyDeviceToHost);
for(int i=0;i<numElements;i++)
{
 cout<<h_C[i]<<endl;
}
return 0;
}
